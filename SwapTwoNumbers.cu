#include "hip/hip_runtime.h"
%%writefile swap.cu
#include <iostream>
__global__ void add( int a, int b, int *c, int *d) {
  *c = b;
  *d = a;
}

int main( void ) {
  int c = 10, d = 11;
  int *dev_c, *dev_d;
  hipMalloc((void**)&dev_c, sizeof(int));
  hipMalloc((void**)&dev_d, sizeof(int));
  add<<<1,1>>>( c, d, dev_c, dev_d );

  hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost );
  hipMemcpy( &d, dev_d, sizeof(int),hipMemcpyDeviceToHost );

  printf( "%d %d", c, d);
  hipFree( dev_c );
  hipFree(dev_d);
  return 0;
}
